#include "hip/hip_runtime.h"
#include "cuda_operations.h"
#include <hip/hip_runtime.h>

// CUDA核函数
__global__ void vectorAddKernel(const float *a, const float *b, float *c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

// 包装函数
void vectorAdd(const float *a, const float *b, float *c, int n) {
    float *d_a, *d_b, *d_c;
    
    // 分配设备内存
    hipMalloc((void**)&d_a, n * sizeof(float));
    hipMalloc((void**)&d_b, n * sizeof(float));
    hipMalloc((void**)&d_c, n * sizeof(float));
    
    // 拷贝数据到设备
    hipMemcpy(d_a, a, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(float), hipMemcpyHostToDevice);
    
    // 启动核函数
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;
    vectorAddKernel<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
    
    // 拷贝结果回主机
    hipMemcpy(c, d_c, n * sizeof(float), hipMemcpyDeviceToHost);
    
    // 释放设备内存
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}